#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "charge.h"
#include <iostream>
#include <fstream>
#include <limits.h>
const double cutoff = 5.0;


void Charge::create_random_numbers
(std::mt19937& generator, int max_value, int total_number, int* random_numbers)
{
    int *permuted_numbers = new int[max_value];
    for(int i = 0; i < max_value; ++i)
    {
        permuted_numbers[i] = i;
    }
    std::uniform_int_distribution<int> rand_int(0, INT_MAX);
    for(int i = 0; i < max_value; ++i)
    {
        int j = rand_int(generator) % (max_value - i) + i;
        int temp = permuted_numbers[i];
        permuted_numbers[i] = permuted_numbers[j];
        permuted_numbers[j] = temp;
    }
    for (int i = 0; i < total_number; ++i)
    {
        random_numbers[i] = permuted_numbers[i];
    }
    delete[] permuted_numbers;
}


void Charge::find_potentials
(
    int number_of_atoms, double box_length[3], int pbc[3],
    std::vector<double>& x,  std::vector<double>& y,  std::vector<double>& z, 
    double* potential
)
{
    double cutoff_square = cutoff * cutoff * xi * xi;
    double xi_factor = -0.5 / (xi * xi);
    double box_length_half[3];
    for (int d = 0; d < 3; ++d) box_length_half[d] = box_length[d] * 0.5;
    for (int n = 0; n < number_of_atoms; ++n) potential[n] = 0.0;
    for (int i = 0; i < Ni; ++i)
    {
        int n1 = impurity_indices[i];
        double x1 = x[n1];
        double y1 = y[n1];
        double z1 = z[n1];
        for (int n2 = 0; n2 < number_of_atoms; ++n2)
        {
            double r12[3];
            r12[0] = x[n2] - x1;
            r12[1] = y[n2] - y1;
            r12[2] = z[n2] - z1;
            double d12_square = 0.0;
            for (int d = 0; d < 3; ++d)
            {
                r12[d] = fabs(r12[d]);
                if (pbc[d] == 1 && r12[d] > box_length_half[d])
                {
                    r12[d] = box_length[d] - r12[d];
                }
                d12_square += r12[d] * r12[d];
            }
            if (d12_square > cutoff_square) continue;
            potential[n2] += impurity_strength[i] * exp(d12_square * xi_factor);
        }
    }
}


void Charge::add_impurities
(
    std::mt19937& generator, int number_of_atoms, double box_length[3],
    int pbc[3],  std::vector<double>& x,  std::vector<double>& y, 
     std::vector<double>& z, double* potential
)
{
    impurity_indices = new int[Ni];
    impurity_strength = new double[Ni];
    create_random_numbers(generator, number_of_atoms, Ni, impurity_indices);
    double W2 = xi * 0.5;
    std::uniform_real_distribution<double> strength(-W2, W2);
    for (int i = 0; i < Ni; ++i)
    {
        impurity_strength[i] = strength(generator);
    }
    find_potentials(number_of_atoms, box_length, pbc, x, y, z, potential);
    delete[] impurity_indices;
    delete[] impurity_strength;
}


