#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "model.h"
#include "vector.h"

#include <fstream>
#include <sstream>
#include <chrono>




Model::Model(std::string input_dir)
{
    this->input_dir = input_dir;
    initialize_parameters();
    initialize_energy();
    if (requires_time)
        initialize_time();
    else
        time_step = 0;
    initialize_neighbor();
    initialize_positions();
    initialize_potential();
    initialize_hopping();
    random_state_real = new real[number_of_atoms]; 
    random_state_imag = new real[number_of_atoms];
 
    // Use higher accuracy clock for the RNG seed
    #ifdef DEBUG
        generator = std::mt19937(12345678);
    #else
       generator = std::mt19937
       (std::chrono::system_clock::now().time_since_epoch().count());
    #endif

    // We only need RNG for random phase generation 
    // so we may use interval [0, 2*PI] right away
    phase_distribution = std::uniform_real_distribution<real>(0, 2 * PI);
    std::cout << "Initialization complete\n" << std::endl;
}




Model::~Model()
{ 
    delete[] energy;
    delete[] time_step;
    delete[] potential;
    delete[] hopping_real;
    delete[] hopping_imag;
    delete[] neighbor_number;
    delete[] neighbor_list;
    delete[] xx;
    delete[] random_state_real;
    delete[] random_state_imag;
    delete[] x;
}




void Model::initialize_state(Vector& random_state)
{
    for (int n = 0; n < number_of_atoms; ++n)
    {  
        real random_phase = get_random_phase();
        random_state_real[n] = cos(random_phase);
        random_state_imag[n] = sin(random_phase);
    }
    random_state.copy_from_host(random_state_real, random_state_imag);
}




void Model::initialize_parameters()
{
    std::string filename = input_dir + "/para.in";
    std::cout << "\nReading " + filename << std::endl;
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout << "Error: cannot open " + filename << std::endl;
        exit(1);
    }
    std::string line;
    
    while (std::getline(input, line))
    {
        std::stringstream ss(line);
        std::string token;
        ss >> token;
        if (token == "") continue;
        if (token == "calculate_vac")
        {
            calculate_vac = true;
        }
        else if (token == "calculate_msd")
        {
            calculate_msd = true;
        }
        else if (token == "number_of_random_vectors")
        {
            ss >> number_of_random_vectors;
        }
        else if (token == "number_of_moments")
        {
            ss >> number_of_moments;
        }
        else if (token == "energy_max")
        {
            ss >> energy_max;
        }
        else
        {
            std::cout << "Unknown identifier in " + input_dir + "/para.in:" << std::endl;
            std::cout << line << std::endl; 
        } 
    }
    input.close();
    
    if (calculate_vac || calculate_msd)
        requires_time = true;
    
    std::cout << "Finished reading " + filename << std::endl; 
    //Verify the used parameters
    std::cout << "- DOS will be calculated" << std::endl;
    if (calculate_vac)
        std::cout << "- VAC will be calculated" << std::endl;
    else
        std::cout << "- VAC is not calculated" << std::endl;
    if (calculate_msd)
        std::cout << "- MSD will be calculated" << std::endl;
    else
        std::cout << "- MSD is not calculated" << std::endl;    
    std::cout << "- Number of random vectors is " 
              << number_of_random_vectors << std::endl; 
    std::cout << "- Number of moments is " 
              << number_of_moments << std::endl;
    std::cout << "- Energy maximum is " << energy_max << std::endl; 
}




void Model::initialize_energy()
{
    std::string filename = input_dir + "/energy.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    input >> number_of_energy_points;
    energy = new real[number_of_energy_points];
    
    for (int n = 0; n < number_of_energy_points; ++n)
    {
        input >> energy[n];
    }
      
    input.close();
    std::cout << "Finished reading " + filename << std::endl;
}




void Model::initialize_time()
{
    std::string filename = input_dir + "/time_step.in";
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    input >> number_of_steps_correlation;
    time_step = new real[number_of_steps_correlation];

    for (int n = 0; n < number_of_steps_correlation; ++n)
    {
        input >> time_step[n];
    }
    
    input.close();
    std::cout << "Finished reading " + filename << std::endl;         
}




void Model::initialize_neighbor()
{
    std::string filename = input_dir + "/neighbor.in";
    std::ifstream input(filename);
    
    if (!input.is_open()) 
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    input >> number_of_atoms >> max_neighbor;
    number_of_pairs = number_of_atoms * max_neighbor;
 
    neighbor_number = new int[number_of_atoms]; 
    neighbor_list = new int[number_of_pairs];

    for (int n = 0; n < number_of_atoms; ++n)
    {
        input >> neighbor_number[n];
        for (int m = 0; m < neighbor_number[n]; ++m)
        {        
            int index = n + m * number_of_atoms;
            input >> neighbor_list[index];
        }
    }
    input.close();
    std::cout << "Finished reading " + filename << std::endl; 
    std::cout << "- Number of atoms is " << number_of_atoms << std::endl;
    std::cout << "- Maximum neighbor number is " << max_neighbor << std::endl;            
}




real Model::get_random_phase()
{
    return phase_distribution(generator);
}




real reduce_distance(real d, real box)
{
    if (d > box/2.0)
        return d-box;
    if (d < -box/2.0)
        return d+box;
    else 
        return d;
}




void Model::initialize_positions()
{
    std::string filename = input_dir + "/position.in";
    std::ifstream input(filename);
    
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }

    input >> box >> volume;   
    x = new real[number_of_atoms];

    for (int i=0; i<number_of_atoms; ++i)
        input >> x[i];
    input.close();
    std::cout << "Finished reading " + filename << std::endl;  
    std::cout << "- Box length along transport direction is " 
              << box << std::endl;
    std::cout << "- System volume is " << volume << std::endl;     
    std::cout << "- Calculating neighbor distances" << std::endl; 
  
    xx = new real[number_of_pairs];    
    for (int n = 0; n < number_of_atoms; ++n)
    {
        for (int m = 0; m < neighbor_number[n]; ++m)
        {        
            int index = n + m * number_of_atoms;
            xx[index] = reduce_distance(x[neighbor_list[index]] - x[n], box);
        }
    }
    std::cout << "- done" << std::endl;
}




void Model::initialize_potential()
{ 
    std::string filename = input_dir + "/potential.in";
    std::ifstream input(filename);
    bool nonzero_potential = true;
    if (!input.is_open())
    {
        std::cout <<"Could not open " + filename << std::endl;
        std::cout << "- Assuming zero onsite potential" << std::endl;
        nonzero_potential = false;
    }

    potential = new real[number_of_atoms];
    
    for (int n = 0; n < number_of_atoms; ++n)
    {
        if (nonzero_potential)
            input >> potential[n];
        else
            potential[n] = 0.0;
    }

    input.close();
    if (nonzero_potential)
        std::cout << "Finished reading " + filename << std::endl; 
}




void Model::initialize_hopping()
{
    std::string filename = input_dir + "/hopping.in";
    std::ifstream input(filename);

    /*
     type == 1 : complex hoppings
     type == 2 : real hoppings
     type == 3 : uniform hoppings (hoppings.in is not read)
    */
    int type = 0;
        
    if (!input.is_open())
    {
        std::cout <<"Could not open " + filename << std::endl;
        type = 3;
    }
    
    std::string first_line;
    

    if (type == 0)
        input >> first_line;
    else
        first_line = ".";
    
    if (first_line == "complex")
    {
        type = 1;
    }
    else if (first_line == "real")
    {
        type = 2;
    }
    else
    {
        type = 3;
        std::cout << "- Assuming uniform hoppings with strength 1" << std::endl;
    }
    
    hopping_real = new real[number_of_pairs]; 
    hopping_imag = new real[number_of_pairs];

    for (int n = 0; n < number_of_atoms; ++n)
    {
        for (int m = 0; m < neighbor_number[n]; ++m)
        {
            int index = n + m * number_of_atoms;
            if (type < 3)
                input >> hopping_real[index];
            else
                hopping_real[index] = 1.0;
            if (type == 1)
                input >> hopping_imag[index];
            else
                hopping_imag[index] = 0.0;
        }
    }
    input.close();
    if (type < 3)
        std::cout << "Finished reading " + filename << std::endl; 
    if (type == 1)
        std::cout << "- Hoppings had imaginary part" << std::endl;
    else if (type == 2)
        std::cout << "- Hoppings were real" << std::endl;
}




