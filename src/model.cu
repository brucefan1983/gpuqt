#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "model.h"
#include "vector.h"
#include <random>
#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>

#define PI 3.141592653589793




Model::Model(std::string input_dir)
{
    // Use higher accuracy clock for the RNG seed
    #ifdef DEBUG
        generator = std::mt19937(12345678);
    #else
        generator = std::mt19937
        (std::chrono::system_clock::now().time_since_epoch().count());
    #endif

    // determine the input directory
    this->input_dir = input_dir;

    // read in para.in
    initialize_parameters();

    // read in energy.in and time_step.in
    initialize_energy();
    if (requires_time)
        initialize_time();
    else
        time_step = 0;

    // initialize the model
    if (use_lattice_model) // use a lattice model
    {
        initialize_lattice_model();
    }
    else // use general inputs to build the model
    {
        initialize_neighbor();
        initialize_positions();
        initialize_potential();
        initialize_hopping();
    }
}




Model::~Model()
{ 
    // other memory will be freed when constructing the Hamiltonian
    delete[] energy;
    delete[] time_step;
}




void Model::initialize_state(Vector& random_state)
{
    std::uniform_real_distribution<real> phase(0, 2 * PI);
    real *random_state_real = new real[number_of_atoms];
    real *random_state_imag = new real[number_of_atoms];

    // spin degeneracy is considered in perform_chebyshev_summation
    if (calculate_spin) // normalize to N/2 to remove spin degeneracy
    {
        for (int n = 0; n < number_of_atoms; n += 2)
        {
            real random_phase = phase(generator);
            random_state_real[n] = cos(random_phase);
            random_state_imag[n] = sin(random_phase);
            random_state_real[n+1] = 0.0;
            random_state_imag[n+1] = 0.0;
        }
    }
    else // normalize to N to keep spin degeneracy
    {
        for (int n = 0; n < number_of_atoms; ++n)
        {
            real random_phase = phase(generator);
            random_state_real[n] = cos(random_phase);
            random_state_imag[n] = sin(random_phase);
        }
    }
    random_state.copy_from_host(random_state_real, random_state_imag);
    delete[] random_state_real;
    delete[] random_state_imag;
}




void Model::print_started_reading(std::string filename)
{
    std::cout << std::endl;
    std::cout << "===========================================================";
    std::cout << std::endl;
    std::cout << "Started reading " + filename << std::endl;
    std::cout << std::endl;
}




void Model::print_finished_reading(std::string filename)
{
    std::cout << std::endl;
    std::cout << "Finished reading " + filename << std::endl;
    std::cout << "===========================================================";
    std::cout << std::endl << std::endl;
}




void Model::initialize_parameters()
{
    std::string filename = input_dir + "/para.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout << "Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    std::string line;
    while (std::getline(input, line))
    {
        std::stringstream ss(line);
        std::string token;
        ss >> token;
        if (token == "") continue;
        if (token == "model")
        {
            ss >> use_lattice_model;
        }
        else if (token == "anderson_disorder")
        {
            has_anderson_disorder = true;
            ss >> anderson_disorder_strength;
        }
        else if (token == "vacancy_disorder")
        {
            has_vacancy_disorder = true;
            ss >> number_of_vacancies;
        }
        else if (token == "calculate_vac")
        {
            calculate_vac = true;
        }
        else if (token == "calculate_msd")
        {
            calculate_msd = true;
        }
        else if (token == "calculate_spin")
        {
            calculate_spin = true;
        }
        else if (token == "number_of_random_vectors")
        {
            ss >> number_of_random_vectors;
        }
        else if (token == "number_of_moments")
        {
            ss >> number_of_moments;
        }
        else if (token == "energy_max")
        {
            ss >> energy_max;
        }
        else
        {
            std::cout << "Unknown identifier in " + input_dir + "/para.in:" 
                      << std::endl;
            std::cout << line << std::endl;
        }
    }
    input.close();
    
    if (calculate_vac || calculate_msd)
        requires_time = true;
    
    //Verify the used parameters (make a seperate function later)
    if (use_lattice_model)
    {
        std::cout << "- Use lattice model" << std::endl;
        if (calculate_spin)
        {
            std::cout << "Error: lattice model does not support "
                      << "spin calculation yet" << std::endl;
            exit(1);
        }
    }
    else
        std::cout << "- Use general model" << std::endl;

    if (has_anderson_disorder)
    {
        std::cout << "- Add Anderson disorder with strength W = "
                  << anderson_disorder_strength << std::endl;
    }

    if (has_vacancy_disorder)
    {
        std::cout << "- Add " << number_of_vacancies
                  << " vacancies" << std::endl;
    }

    std::cout << "- DOS will be calculated" << std::endl;

    if (calculate_vac)
        std::cout << "- VAC will be calculated" << std::endl;
    else
        std::cout << "- VAC will not be calculated" << std::endl;

    if (calculate_msd)
        std::cout << "- MSD will be calculated" << std::endl;
    else
        std::cout << "- MSD will not be calculated" << std::endl;

    if (calculate_spin)
        std::cout << "- spin polarization will be calculated" << std::endl;
    else
        std::cout << "- spin polarization will not be calculated" << std::endl;

    if (calculate_spin && calculate_vac)
    {
        std::cout << "Error: spin and VAC cannot be calculated together"
                  << std::endl;
        exit(1);
    }

    if (calculate_spin && calculate_msd)
    {
        std::cout << "Error: spin and MSD cannot be calculated together"
                  << std::endl;
        exit(1);
    }

    std::cout << "- Number of random vectors is "
              << number_of_random_vectors << std::endl;
    std::cout << "- Number of moments is "
              << number_of_moments << std::endl;
    std::cout << "- Energy maximum is " << energy_max << std::endl;

    print_finished_reading(filename);
}




void Model::initialize_energy()
{
    std::string filename = input_dir + "/energy.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }

    print_started_reading(filename);

    input >> number_of_energy_points;
    std::cout << "- number of energy points = "
              << number_of_energy_points 
              << std::endl;
    energy = new real[number_of_energy_points];

    for (int n = 0; n < number_of_energy_points; ++n)
    {
        input >> energy[n];
    }

    input.close();

    print_finished_reading(filename);
}




void Model::initialize_time()
{
    std::string filename = input_dir + "/time_step.in";
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    input >> number_of_steps_correlation;
    time_step = new real[number_of_steps_correlation];

    for (int n = 0; n < number_of_steps_correlation; ++n)
    {
        input >> time_step[n];
    }

    input.close();
    print_finished_reading(filename);
}




