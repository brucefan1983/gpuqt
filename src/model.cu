#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "model.h"
#include "vector.h"
#include <random>
#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>
#include <limits.h>

#define PI 3.141592653589793




Model::Model(std::string input_dir)
{
    // Use higher accuracy clock for the RNG seed
    #ifdef DEBUG
        generator = std::mt19937(12345678);
    #else
        generator = std::mt19937
        (std::chrono::system_clock::now().time_since_epoch().count());
    #endif

    // determine the input directory
    this->input_dir = input_dir;

    // read in para.in
    initialize_parameters();

    // read in energy.in and time_step.in
    initialize_energy();
    if (requires_time)
        initialize_time();
    else
        time_step = 0;

    // initialize the model
    if (use_lattice_model) // use a lattice model
    {
        initialize_lattice_model();
    }
    else // use general inputs to build the model
    {
        initialize_neighbor();
        initialize_positions();
        initialize_potential();
        initialize_hopping();
    }
}




Model::~Model()
{ 
    // other memory will be freed when constructing the Hamiltonian
    delete[] energy;
    delete[] time_step;
}




void Model::initialize_state(Vector& random_state)
{
    std::uniform_real_distribution<real> phase(0, 2 * PI);
    real *random_state_real = new real[number_of_atoms];
    real *random_state_imag = new real[number_of_atoms];
    for (int n = 0; n < number_of_atoms; ++n)
    {
        real random_phase = phase(generator);
        random_state_real[n] = cos(random_phase);
        random_state_imag[n] = sin(random_phase);
    }
    random_state.copy_from_host(random_state_real, random_state_imag);
    delete[] random_state_real;
    delete[] random_state_imag;
}




static void print_started_reading(std::string filename)
{
    std::cout << std::endl;
    std::cout << "===========================================================";
    std::cout << std::endl;
    std::cout << "Started reading " + filename << std::endl;
    std::cout << std::endl;
}




static void print_finished_reading(std::string filename)
{
    std::cout << std::endl;
    std::cout << "Finished reading " + filename << std::endl;
    std::cout << "===========================================================";
    std::cout << std::endl << std::endl;
}




void Model::initialize_parameters()
{
    std::string filename = input_dir + "/para.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout << "Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    std::string line;
    while (std::getline(input, line))
    {
        std::stringstream ss(line);
        std::string token;
        ss >> token;
        if (token == "") continue;
        if (token == "model")
        {
            ss >> use_lattice_model;
        }
        else if (token == "anderson_disorder")
        {
            has_anderson_disorder = true;
            ss >> anderson_disorder_strength;
        }
        else if (token == "vacancy_disorder")
        {
            has_vacancy_disorder = true;
            ss >> number_of_vacancies;
        }
        else if (token == "calculate_vac")
        {
            calculate_vac = true;
        }
        else if (token == "calculate_msd")
        {
            calculate_msd = true;
        }
        else if (token == "number_of_random_vectors")
        {
            ss >> number_of_random_vectors;
        }
        else if (token == "number_of_moments")
        {
            ss >> number_of_moments;
        }
        else if (token == "energy_max")
        {
            ss >> energy_max;
        }
        else
        {
            std::cout << "Unknown identifier in " + input_dir + "/para.in:" 
                      << std::endl;
            std::cout << line << std::endl;
        }
    }
    input.close();
    
    if (calculate_vac || calculate_msd)
        requires_time = true;
    
    //Verify the used parameters (make a seperate function later)
    if (use_lattice_model)
        std::cout << "- Use lattice model" << std::endl;
    else
        std::cout << "- Use general model" << std::endl;

    if (has_anderson_disorder)
    {
        std::cout << "- Add Anderson disorder with strength W = "
                  << anderson_disorder_strength << std::endl;
    }

    if (has_vacancy_disorder)
    {
        std::cout << "- Add " << number_of_vacancies
                  << " vacancies" << std::endl;
    }

    std::cout << "- DOS will be calculated" << std::endl;
    if (calculate_vac)
        std::cout << "- VAC will be calculated" << std::endl;
    else
        std::cout << "- VAC will not be calculated" << std::endl;
    if (calculate_msd)
        std::cout << "- MSD will be calculated" << std::endl;
    else
        std::cout << "- MSD will not be calculated" << std::endl;
    std::cout << "- Number of random vectors is "
              << number_of_random_vectors << std::endl;
    std::cout << "- Number of moments is "
              << number_of_moments << std::endl;
    std::cout << "- Energy maximum is " << energy_max << std::endl;

    print_finished_reading(filename);
}




void Model::initialize_energy()
{
    std::string filename = input_dir + "/energy.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }

    print_started_reading(filename);

    input >> number_of_energy_points;
    std::cout << "- number of energy points = "
              << number_of_energy_points 
              << std::endl;
    energy = new real[number_of_energy_points];

    for (int n = 0; n < number_of_energy_points; ++n)
    {
        input >> energy[n];
    }

    input.close();

    print_finished_reading(filename);
}




void Model::initialize_time()
{
    std::string filename = input_dir + "/time_step.in";
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    input >> number_of_steps_correlation;
    time_step = new real[number_of_steps_correlation];

    for (int n = 0; n < number_of_steps_correlation; ++n)
    {
        input >> time_step[n];
    }

    input.close();
    print_finished_reading(filename);
}




void Model::initialize_neighbor()
{
    std::string filename = input_dir + "/neighbor.in";
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    input >> number_of_atoms >> max_neighbor;
    number_of_pairs = number_of_atoms * max_neighbor;

    neighbor_number = new int[number_of_atoms];
    neighbor_list = new int[number_of_pairs];

    for (int n = 0; n < number_of_atoms; ++n)
    {
        input >> neighbor_number[n];
        for (int m = 0; m < neighbor_number[n]; ++m)
        {        
            int index = n * max_neighbor + m;
            input >> neighbor_list[index];
        }
    }

    input.close();

    std::cout << "- Number of atoms is " << number_of_atoms << std::endl;
    std::cout << "- Maximum neighbor number is " << max_neighbor << std::endl;
    print_finished_reading(filename);
}




real reduce_distance(real d, real box)
{
    if (d > box/2.0)
        return d-box;
    if (d < -box/2.0)
        return d+box;
    else
        return d;
}




void Model::initialize_positions()
{
    std::string filename = input_dir + "/position.in";
    std::ifstream input(filename);
    
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    real box;
    input >> box >> volume;
    real *x = new real[number_of_atoms];

    for (int i=0; i<number_of_atoms; ++i)
        input >> x[i];
    input.close();
  
    std::cout << "- Box length along transport direction is "
              << box << std::endl;
    std::cout << "- System volume is " << volume << std::endl;

    xx = new real[number_of_pairs];
    for (int n = 0; n < number_of_atoms; ++n)
    {
        for (int m = 0; m < neighbor_number[n]; ++m)
        {
            int index = n * max_neighbor + m;
            xx[index] = reduce_distance(x[neighbor_list[index]] - x[n], box);
        }
    }

    delete[] x;
    print_finished_reading(filename);
}




void Model::initialize_potential()
{
    std::string filename = input_dir + "/potential.in";
    print_started_reading(filename);

    std::ifstream input(filename);
    bool nonzero_potential = true;
    if (!input.is_open())
    {
        std::cout <<"- Could not open " + filename << std::endl;
        std::cout << "- Assuming zero onsite potential" << std::endl;
        nonzero_potential = false;
    }

    potential = new real[number_of_atoms];

    for (int n = 0; n < number_of_atoms; ++n)
    {
        if (nonzero_potential)
            input >> potential[n];
        else
            potential[n] = 0.0;
    }

    input.close();

    print_finished_reading(filename);
}




void Model::initialize_hopping()
{
    std::string filename = input_dir + "/hopping.in";
    print_started_reading(filename);
    std::ifstream input(filename);

    /*
     type == 1 : complex hoppings
     type == 2 : real hoppings
     type == 3 : uniform hoppings (hoppings.in is not read)
    */
    int type = 0;

    if (!input.is_open())
    {
        type = 3;
        std::cout <<"- Could not open " + filename << std::endl;
        std::cout << "- Assuming uniform hoppings with strength -1"
                  << std::endl;
    }
    else
    {
        std::string first_line;
        input >> first_line;
        if (first_line == "complex")
        {
            type = 1;
            std::cout << "- Hoppings have imaginary part" << std::endl;
        }
        else if (first_line == "real")
        {
            type = 2;
            std::cout << "- Hoppings are real" << std::endl;
        }
        else
        {
            std::cout << "- Hoppings can only be real or complex"
                      << std::endl;
            exit(1);
        }
    }

    hopping_real = new real[number_of_pairs];
    hopping_imag = new real[number_of_pairs];
    for (int n = 0; n < number_of_atoms; ++n)
    {
        for (int m = 0; m < neighbor_number[n]; ++m)
        {
            int index = n * max_neighbor + m;
            if (type < 3)
                input >> hopping_real[index];
            else
                hopping_real[index] = -1.0;
            if (type == 1)
                input >> hopping_imag[index];
            else
                hopping_imag[index] = 0.0;
        }
    }
    input.close();

    print_finished_reading(filename);
}




void Model::add_anderson_disorder()
{
    potential = new real[number_of_atoms];
    real W2 = anderson_disorder_strength * 0.5;
    std::uniform_real_distribution<real> on_site_potential(-W2, W2);
    for (int n = 0; n < number_of_atoms; ++n)
    {
        if (has_anderson_disorder)
            potential[n] = on_site_potential(generator);
        else
            potential[n] = 0.0;
    }
}




void Model::create_random_numbers
(int max_value, int total_number, int* random_numbers)
{
    int *permuted_numbers = new int[max_value];
    for(int i = 0; i < max_value; ++i)
    {
        permuted_numbers[i] = i;
    }
    std::uniform_int_distribution<int> rand_int(0, INT_MAX);
    for(int i = 0; i < max_value; ++i)
    {
        int j = rand_int(generator) % (max_value - i) + i;
        int temp = permuted_numbers[i];
        permuted_numbers[i] = permuted_numbers[j];
        permuted_numbers[j] = temp;
    }
    for (int i = 0; i < total_number; ++i)
    {
        random_numbers[i] = permuted_numbers[i];
    }
    delete[] permuted_numbers;
}




void Model::specify_vacancies
(int *is_vacancy, int number_of_atoms_pristine)
{
    int *vacancy_indices = new int[number_of_vacancies];
    create_random_numbers
    (number_of_atoms_pristine, number_of_vacancies, vacancy_indices);

    for (int n = 0; n < number_of_atoms_pristine; ++n)
    {
        is_vacancy[n] = 0;
    }
    for (int n = 0; n < number_of_vacancies; ++n)
    {
        is_vacancy[vacancy_indices[n]] = 1;
    }
    delete[] vacancy_indices;
}




void Model::find_new_atom_index
(int *is_vacancy, int *new_atom_index, int number_of_atoms_pristine)
{
    int count = 0;
    for (int n = 0; n < number_of_atoms_pristine; ++n)
    {
        if (is_vacancy[n] == 0)
        {
            new_atom_index[n] = count;
            ++count;
        }
    }
}




void Model::add_vacancies()
{
    // copy some data
    int *neighbor_number_pristine = new int[number_of_atoms];
    int *neighbor_list_pristine = new int[number_of_pairs];
    real *hopping_real_pristine = new real[number_of_pairs];
    real *hopping_imag_pristine = new real[number_of_pairs];
    real *xx_pristine = new real[number_of_pairs];

    for (int n = 0; n < number_of_atoms; ++n)
    {
        neighbor_number_pristine[n] = neighbor_number[n];
    }   
    for (int m = 0; m < number_of_pairs; ++m)
    {
        neighbor_list_pristine[m] = neighbor_list[m];
        hopping_real_pristine[m] = hopping_real[m];
        hopping_imag_pristine[m] = hopping_imag[m];
        xx_pristine[m] = xx[m];
    }

    // change parameters
    int number_of_atoms_pristine = number_of_atoms;
    number_of_atoms = number_of_atoms_pristine - number_of_vacancies;
    number_of_pairs = number_of_atoms * max_neighbor;

    // delete old memory
    delete[] neighbor_number;
    delete[] neighbor_list;
    delete[] hopping_real;
    delete[] hopping_imag;
    delete[] xx;

    // allocate new memory
    neighbor_number = new int[number_of_atoms];
    neighbor_list = new int[number_of_pairs];
    hopping_real = new real[number_of_pairs];
    hopping_imag = new real[number_of_pairs];
    xx = new real[number_of_pairs];

    // specify the distribution of the vacancies
    int *is_vacancy = new int[number_of_atoms_pristine];
    specify_vacancies(is_vacancy, number_of_atoms_pristine);

    // find the new indices of the atoms
    int *new_atom_index = new int[number_of_atoms_pristine];
    find_new_atom_index(is_vacancy, new_atom_index, number_of_atoms_pristine);

    // get the new neighbor structure and related data
    int count_atom = 0;
    for (int n = 0; n < number_of_atoms_pristine; ++n)
    {
        if (is_vacancy[n] == 0)
        {
            int count_neighbor = 0;
            for (int m = 0; m < neighbor_number_pristine[n]; ++m)
            {
                int index_old = n * max_neighbor + m;
                int k = neighbor_list_pristine[index_old];
                if (is_vacancy[k] == 0)
                {
                    int index_new = count_atom * max_neighbor + count_neighbor;
                    neighbor_list[index_new] = new_atom_index[k];
                    hopping_real[index_new] = hopping_real_pristine[index_old];
                    hopping_imag[index_new] = hopping_imag_pristine[index_old];
                    xx[index_new] = xx_pristine[index_old];
                    ++count_neighbor;
                }
            }
            neighbor_number[count_atom] = count_neighbor;
            ++count_atom;
        }
    }

    // free memory
    delete[] neighbor_number_pristine;
    delete[] neighbor_list_pristine;
    delete[] hopping_real_pristine;
    delete[] hopping_imag_pristine;
    delete[] xx_pristine;
    delete[] is_vacancy;  
    delete[] new_atom_index;
}




static int find_index
(int nx, int ny, int nz, int Nx, int Ny, int Nz, int m, int N_orbital)
{
    if (nx < 0) nx += Nx;
    if (nx >= Nx) nx -= Nx;
    if (ny < 0) ny += Ny;
    if (ny >= Ny) ny -= Ny;
    if (nz < 0) nz += Nz;
    if (nz >= Nz) nz -= Nz;
    return ((nx * Ny + ny) * Nz + nz) * N_orbital + m;
}




void Model::initialize_lattice_model()
{
    std::string filename = input_dir + "/lattice.in";
    print_started_reading(filename);
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Could not open " + filename << std::endl;
        exit(1);
    }
 
    int N_orbital;
    int transport_direction;
    int N_cell[3];
    int pbc[3];
    real box[3];
    real lattice_constant[3];

    input >> N_cell[0] >> N_cell[1] >> N_cell[2];
    std::cout << "number of cells  = " 
         << N_cell[0] << " " << N_cell[1] << " " << N_cell[2] << std::endl;

    input >> pbc[0] >> pbc[1] >> pbc[2] >> transport_direction;
    std::cout << "pbc = " << pbc[0] << " " << pbc[1] << " " << pbc[2] 
              << std::endl;
    std::cout << "transport direction = " << transport_direction << std::endl;

    if (pbc[transport_direction] != 1)
    {
        std::cout << "Error: transport direction must be periodic" << std::endl;
        exit(1);
    }

    input >> lattice_constant[0] >> lattice_constant[1] >> lattice_constant[2];
    std::cout << "lattice constant = " 
         << lattice_constant[0] << " "
         << lattice_constant[1] << " "
         << lattice_constant[2] << " "
         << std::endl;
    for (int d = 0; d < 3; ++d)
        box[d] = lattice_constant[d] * N_cell[d];
    volume = box[0] * box[1] * box[2];
    std::cout << "box = " << box[0] << " " << box[1] << " " << box[2] << " "
              << std::endl;

    input >> N_orbital >> max_neighbor;
    std::cout << "nnumber of orbitals per cell = " << N_orbital << std::endl;
    std::cout << "maximum number of hoppings per orbital = " << max_neighbor
              << std::endl;
    number_of_atoms = N_orbital * N_cell[0] * N_cell[1] * N_cell[2];
    std::cout << "number_of_atoms = " << number_of_atoms << std::endl;

    number_of_pairs = number_of_atoms * max_neighbor;
    neighbor_number = new int[number_of_atoms];
    neighbor_list = new int [number_of_pairs];
    hopping_real = new real[number_of_pairs];
    hopping_imag = new real[number_of_pairs];
    xx = new real[number_of_pairs];

    std::vector<real> x_cell;
    x_cell.resize(N_orbital);
    int number_of_hoppings_per_cell = N_orbital * max_neighbor;
    std::vector<std::vector<int>> hopping_data;
    hopping_data.assign(6, std::vector<int>(number_of_hoppings_per_cell, 0));

    std::cout << std::endl << "orbital\tx" << std::endl;
    for (int n = 0; n < N_orbital; ++n)
    {
        input >> x_cell[n];
        std::cout << n << "\t" << x_cell[n] << std::endl;
    }

    std::vector<int> number_of_hoppings;
    number_of_hoppings.resize(N_orbital);
    for (int m = 0; m < N_orbital; m++)
    {
        input >> number_of_hoppings[m];
        std::cout << std::endl << "number_of_hoppings for orbital " << m
                  << " = " << number_of_hoppings[m] << std::endl;

        for (int n = 0; n < number_of_hoppings[m]; ++n)
        {
            int nx, ny, nz, m_neighbor;
            real hopping_real, hopping_imag;
            input >> nx >> ny >> nz >> m_neighbor >> hopping_real
                  >> hopping_imag;

            hopping_data[0][m*max_neighbor+n] = nx;
            hopping_data[1][m*max_neighbor+n] = ny;
            hopping_data[2][m*max_neighbor+n] = nz;
            hopping_data[3][m*max_neighbor+n] = m_neighbor;
            hopping_data[4][m*max_neighbor+n] = hopping_real;
            hopping_data[5][m*max_neighbor+n] = hopping_imag;

            std::cout << "H(0,0,0," << m << "; " 
                 << nx << "," << ny << "," << nz << "," << m_neighbor << ") = "
                 << hopping_real << " + i " << hopping_imag << std::endl;
        }
    }

    for (int nx1 = 0; nx1 < N_cell[0]; ++nx1)
    {
        for (int ny1 = 0; ny1 < N_cell[1]; ++ny1)
        {  
            for (int nz1 = 0; nz1 < N_cell[2]; ++nz1)
            {
                for (int m = 0; m < N_orbital; ++m)
                {
                    int n1 = find_index
                    (
                        nx1, ny1, nz1, N_cell[0], N_cell[1], N_cell[2], 
                        m, N_orbital
                    );

                    int count = 0;
                    for (int i = 0; i < number_of_hoppings[m]; ++i)
                    {
                        int neighbor_index = n1 * max_neighbor + count;
                        int k = m*max_neighbor+i;

                        int nx2 = hopping_data[0][k] + nx1;
                        int ny2 = hopping_data[1][k] + ny1;
                        int nz2 = hopping_data[2][k] + nz1;
                        bool skip_x = !pbc[0] && (nx2 < 0 || nx2 >= N_cell[0]);
                        bool skip_y = !pbc[1] && (ny2 < 0 || ny2 >= N_cell[1]);
                        bool skip_z = !pbc[2] && (nz2 < 0 || nz2 >= N_cell[2]);
                        if (skip_x || skip_y || skip_z) continue;

                        neighbor_list[neighbor_index] = find_index
                        (
                            nx2, ny2, nz2, N_cell[0], N_cell[1], N_cell[2], 
                            hopping_data[3][k], N_orbital
                        );

                        real x12 = lattice_constant[transport_direction]
                                 * hopping_data[transport_direction][k];
                        x12 += x_cell[hopping_data[3][k]] - x_cell[m];
                        xx[neighbor_index] = x12;

                        hopping_real[neighbor_index] = hopping_data[4][k];
                        hopping_imag[neighbor_index] = hopping_data[5][k];

                        ++count;
                    } 
                    neighbor_number[n1] = count;
                }
            }
        }
    }

    if (has_vacancy_disorder && number_of_vacancies > 0)
    {
        add_vacancies();
    }
    add_anderson_disorder();
    print_finished_reading(filename);
}




