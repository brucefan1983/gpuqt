#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "model.h"
#include "vector.h"

#include <fstream>
#include <sstream>
#include <chrono>



Model::Model(std::string input_dir)
{
    // Use higher accuracy clock for the RNG seed
    #ifdef DEBUG
        generator = std::mt19937(12345678);
    #else
        generator = std::mt19937
        (std::chrono::system_clock::now().time_since_epoch().count());
    #endif

    // determine the input directory
    this->input_dir = input_dir;

    // read in para.in
    initialize_parameters();

    // read in energy.in and time_step.in
    initialize_energy();
    if (requires_time)
        initialize_time();
    else
        time_step = 0;

    // initialize the model
    if (use_lattice_model) // use a lattice model
    {
        initialize_lattice_model();
        // then consider disorders
        if (has_anderson_disorder)
            add_anderson_disorder();
    }
    else // use general inputs to build the model
    {
        initialize_neighbor();
        initialize_positions();
        initialize_potential();
        initialize_hopping();
    }
}




Model::~Model()
{ 
    delete[] energy;
    delete[] time_step;
    delete[] potential;
    delete[] hopping_real;
    delete[] hopping_imag;
    delete[] neighbor_number;
    delete[] neighbor_list;
    delete[] xx;
}




void Model::initialize_state(Vector& random_state)
{
    std::uniform_real_distribution<real> phase(0, 2 * PI);
    real *random_state_real = new real[number_of_atoms]; 
    real *random_state_imag = new real[number_of_atoms];
    for (int n = 0; n < number_of_atoms; ++n)
    {  
        real random_phase = phase(generator);
        random_state_real[n] = cos(random_phase);
        random_state_imag[n] = sin(random_phase);
    }
    random_state.copy_from_host(random_state_real, random_state_imag);
    delete[] random_state_real;
    delete[] random_state_imag;
}




static void print_started_reading(std::string filename)
{
    std::cout << std::endl;
    std::cout << "===========================================================";
    std::cout << std::endl;
    std::cout << "Started reading " + filename << std::endl;
    std::cout << std::endl;
}




static void print_finished_reading(std::string filename)
{
    std::cout << std::endl;
    std::cout << "Finished reading " + filename << std::endl; 
    std::cout << "===========================================================";
    std::cout << std::endl << std::endl;
}




void Model::initialize_parameters()
{
    std::string filename = input_dir + "/para.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout << "Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    std::string line;
    while (std::getline(input, line))
    {
        std::stringstream ss(line);
        std::string token;
        ss >> token;
        if (token == "") continue;
        if (token == "model")
        {
            ss >> use_lattice_model;
        }
        else if (token == "anderson_disorder")
        {
            has_anderson_disorder = true;
            ss >> anderson_disorder_strength;
        }
        else if (token == "calculate_vac")
        {
            calculate_vac = true;
        }
        else if (token == "calculate_msd")
        {
            calculate_msd = true;
        }
        else if (token == "number_of_random_vectors")
        {
            ss >> number_of_random_vectors;
        }
        else if (token == "number_of_moments")
        {
            ss >> number_of_moments;
        }
        else if (token == "energy_max")
        {
            ss >> energy_max;
        }
        else
        {
            std::cout << "Unknown identifier in " + input_dir + "/para.in:" << std::endl;
            std::cout << line << std::endl; 
        } 
    }
    input.close();
    
    if (calculate_vac || calculate_msd)
        requires_time = true;
    
    //Verify the used parameters (make a seperate function later)
    if (use_lattice_model) 
        std::cout << "- Use lattice model" << std::endl;
    else
        std::cout << "- Use general model" << std::endl;

    if (has_anderson_disorder)
    {
        std::cout << "- Add Anderson disorder with strength W = " 
                  << anderson_disorder_strength << std::endl;
    }

    std::cout << "- DOS will be calculated" << std::endl;
    if (calculate_vac)
        std::cout << "- VAC will be calculated" << std::endl;
    else
        std::cout << "- VAC will not be calculated" << std::endl;
    if (calculate_msd)
        std::cout << "- MSD will be calculated" << std::endl;
    else
        std::cout << "- MSD will not be calculated" << std::endl;    
    std::cout << "- Number of random vectors is " 
              << number_of_random_vectors << std::endl; 
    std::cout << "- Number of moments is " 
              << number_of_moments << std::endl;
    std::cout << "- Energy maximum is " << energy_max << std::endl; 

    print_finished_reading(filename);
}




void Model::initialize_energy()
{
    std::string filename = input_dir + "/energy.in";
    std::ifstream input(filename);
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }

    print_started_reading(filename);

    input >> number_of_energy_points;
    std::cout << "- number of energy points = " 
              << number_of_energy_points 
              << std::endl;
    energy = new real[number_of_energy_points];
    
    for (int n = 0; n < number_of_energy_points; ++n)
    {
        input >> energy[n];
    }
      
    input.close();

    print_finished_reading(filename);
}




void Model::initialize_time()
{
    std::string filename = input_dir + "/time_step.in";
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    input >> number_of_steps_correlation;
    time_step = new real[number_of_steps_correlation];

    for (int n = 0; n < number_of_steps_correlation; ++n)
    {
        input >> time_step[n];
    }
    
    input.close();
    print_finished_reading(filename);         
}




void Model::initialize_neighbor()
{
    std::string filename = input_dir + "/neighbor.in";
    std::ifstream input(filename);
    
    if (!input.is_open()) 
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    input >> number_of_atoms >> max_neighbor;
    number_of_pairs = number_of_atoms * max_neighbor;
 
    neighbor_number = new int[number_of_atoms]; 
    neighbor_list = new int[number_of_pairs];

    for (int n = 0; n < number_of_atoms; ++n)
    {
        input >> neighbor_number[n];
        for (int m = 0; m < neighbor_number[n]; ++m)
        {        
            int index = n + m * number_of_atoms;
            input >> neighbor_list[index];
        }
    }

    input.close();
 
    std::cout << "- Number of atoms is " << number_of_atoms << std::endl;
    std::cout << "- Maximum neighbor number is " << max_neighbor << std::endl;
    print_finished_reading(filename);            
}




real reduce_distance(real d, real box)
{
    if (d > box/2.0)
        return d-box;
    if (d < -box/2.0)
        return d+box;
    else 
        return d;
}




void Model::initialize_positions()
{
    std::string filename = input_dir + "/position.in";
    std::ifstream input(filename);
    
    if (!input.is_open())
    {
        std::cout <<"Error: cannot open " + filename << std::endl;
        exit(1);
    }
    print_started_reading(filename);

    real box;
    input >> box >> volume;   
    real *x = new real[number_of_atoms];

    for (int i=0; i<number_of_atoms; ++i)
        input >> x[i];
    input.close();
  
    std::cout << "- Box length along transport direction is " 
              << box << std::endl;
    std::cout << "- System volume is " << volume << std::endl;      
  
    xx = new real[number_of_pairs];    
    for (int n = 0; n < number_of_atoms; ++n)
    {
        for (int m = 0; m < neighbor_number[n]; ++m)
        {        
            int index = n + m * number_of_atoms;
            xx[index] = reduce_distance(x[neighbor_list[index]] - x[n], box);
        }
    }

    delete[] x;
    print_finished_reading(filename);
}




void Model::initialize_potential()
{ 
    std::string filename = input_dir + "/potential.in";
    print_started_reading(filename);

    std::ifstream input(filename);
    bool nonzero_potential = true;
    if (!input.is_open())
    {
        std::cout <<"- Could not open " + filename << std::endl;
        std::cout << "- Assuming zero onsite potential" << std::endl;
        nonzero_potential = false;
    }

    potential = new real[number_of_atoms];
    
    for (int n = 0; n < number_of_atoms; ++n)
    {
        if (nonzero_potential)
            input >> potential[n];
        else
            potential[n] = 0.0;
    }

    input.close();
 
    print_finished_reading(filename);
}




void Model::add_anderson_disorder()
{ 
    potential = new real[number_of_atoms];
    real W2 = anderson_disorder_strength * 0.5;
    std::uniform_real_distribution<real> on_site_potential(-W2, W2);
    for (int n = 0; n < number_of_atoms; ++n)
    {
        potential[n] = on_site_potential(generator);
    }
}




void Model::initialize_hopping()
{
    std::string filename = input_dir + "/hopping.in";
    print_started_reading(filename);
    std::ifstream input(filename);

    /*
     type == 1 : complex hoppings
     type == 2 : real hoppings
     type == 3 : uniform hoppings (hoppings.in is not read)
    */
    int type = 0;
        
    if (!input.is_open())
    {
        type = 3;
        std::cout <<"- Could not open " + filename << std::endl;
        std::cout << "- Assuming uniform hoppings with strength -1" << std::endl;
    }
    else
    {
        std::string first_line;
        input >> first_line;
        if (first_line == "complex")
        {
            type = 1;
            std::cout << "- Hoppings have imaginary part" << std::endl;
        }
        else if (first_line == "real")
        {
            type = 2;
            std::cout << "- Hoppings are real" << std::endl;
        }
        else
        {
            std::cout << "- Hoppings can only be real or complex"
                      << std::endl;
            exit(1);
        }
    }
    
    hopping_real = new real[number_of_pairs]; 
    hopping_imag = new real[number_of_pairs];
    for (int n = 0; n < number_of_atoms; ++n)
    {
        for (int m = 0; m < neighbor_number[n]; ++m)
        {
            int index = n + m * number_of_atoms;
            if (type < 3)
                input >> hopping_real[index];
            else
                hopping_real[index] = -1.0;
            if (type == 1)
                input >> hopping_imag[index];
            else
                hopping_imag[index] = 0.0;
        }
    }
    input.close();

    print_finished_reading(filename);
}




static int find_index
(int nx, int ny, int nz, int Nx, int Ny, int Nz, int m, int N_orbital)
{ 
    if (nx < 0) nx += Nx;
    if (nx >= Nx) nx -= Nx;
    if (ny < 0) ny += Ny;
    if (ny >= Ny) ny -= Ny;
    if (nz < 0) nz += Nz;
    if (nz >= Nz) nz -= Nz;
    return ((nx * Ny + ny) * Nz + nz) * N_orbital + m;
}




void Model::initialize_lattice_model()
{
    std::string filename = input_dir + "/lattice.in";
    print_started_reading(filename);
    std::ifstream input(filename);

    if (!input.is_open())
    {
        std::cout <<"Could not open " + filename << std::endl;
        exit(1);
    }
 
    int N_orbital;
    int transport_direction;
    int N_cell[3];
    int pbc[3];
    real box[3];
    real lattice_constant[3];

    input >> N_cell[0] >> N_cell[1] >> N_cell[2];
    std::cout << "number of cells  = " 
         << N_cell[0] << " " << N_cell[1] << " " << N_cell[2] << std::endl;

    input >> pbc[0] >> pbc[1] >> pbc[2] >> transport_direction;
    std::cout << "pbc = " << pbc[0] << " " << pbc[1] << " " << pbc[2] 
              << std::endl;
    std::cout << "transport direction = " << transport_direction << std::endl;

    if (pbc[transport_direction] != 1)
    {
        std::cout << "Error: transport direction must be periodic" << std::endl;
        exit(1);
    }

    input >> lattice_constant[0] >> lattice_constant[1] >> lattice_constant[2];
    std::cout << "lattice constant = " 
         << lattice_constant[0] << " "
         << lattice_constant[1] << " "
         << lattice_constant[2] << " "
         << std::endl;
    for (int d = 0; d < 3; ++d)
        box[d] = lattice_constant[d] * N_cell[d];
    volume = box[0] * box[1] * box[2];
    std::cout << "box = " << box[0] << " " << box[1] << " " << box[2] << " "
         << std::endl;

    input >> N_orbital >> max_neighbor;
    std::cout << "nnumber of orbitals per cell = " << N_orbital << std::endl;
    std::cout << "maximum number of hoppings per orbital = " << max_neighbor << std::endl;
    number_of_atoms = N_orbital * N_cell[0] * N_cell[1] * N_cell[2];
    std::cout << "number_of_atoms = " << number_of_atoms << std::endl;

    number_of_pairs = number_of_atoms * max_neighbor;
    neighbor_number = new int[number_of_atoms];
    neighbor_list = new int [number_of_pairs];
    hopping_real = new real[number_of_pairs];
    hopping_imag = new real[number_of_pairs];
    xx = new real[number_of_pairs];

    std::vector<real> x_cell;
    x_cell.resize(N_orbital);
    int number_of_hoppings_per_cell = N_orbital * max_neighbor;
    std::vector<std::vector<int>> hopping_data;
    hopping_data.assign(6, std::vector<int>(number_of_hoppings_per_cell, 0));
 
    std::cout << std::endl << "orbital\tx" << std::endl;
    for (int n = 0; n < N_orbital; ++n)
    {
        input >> x_cell[n];
        std::cout << n << "\t" << x_cell[n] << std::endl;
    }

    std::vector<int> number_of_hoppings;
    number_of_hoppings.resize(N_orbital);
    for (int m = 0; m < N_orbital; m++)
    {
        input >> number_of_hoppings[m];
        std::cout << std::endl << "number_of_hoppings for orbital " << m << " = " 
             << number_of_hoppings[m] << std::endl;

        for (int n = 0; n < number_of_hoppings[m]; ++n)
        {
            int nx, ny, nz, m_neighbor;
            real hopping_real, hopping_imag;
            input >> nx >> ny >> nz >> m_neighbor >> hopping_real 
                  >> hopping_imag;

            hopping_data[0][m*max_neighbor+n] = nx;
            hopping_data[1][m*max_neighbor+n] = ny;
            hopping_data[2][m*max_neighbor+n] = nz;
            hopping_data[3][m*max_neighbor+n] = m_neighbor;
            hopping_data[4][m*max_neighbor+n] = hopping_real;
            hopping_data[5][m*max_neighbor+n] = hopping_imag;

            std::cout << "H(0,0,0," << m << "; " 
                 << nx << "," << ny << "," << nz << "," << m_neighbor << ") = "
                 << hopping_real << " + i " << hopping_imag << std::endl;
        }
    }  


    for (int nx1 = 0; nx1 < N_cell[0]; ++nx1)
    {
        for (int ny1 = 0; ny1 < N_cell[1]; ++ny1)
        {  
            for (int nz1 = 0; nz1 < N_cell[2]; ++nz1)
            {
                for (int m = 0; m < N_orbital; ++m)
                {
                    int n1 = find_index
                    (
                        nx1, ny1, nz1, N_cell[0], N_cell[1], N_cell[2], 
                        m, N_orbital
                    );

                    int count = 0;
                    for (int i = 0; i < number_of_hoppings[m]; ++i)
                    {
                        int neighbor_index = n1 + count * number_of_atoms;
                        int k = m*max_neighbor+i;

                        int nx2 = hopping_data[0][k] + nx1;
                        int ny2 = hopping_data[1][k] + ny1;
                        int nz2 = hopping_data[2][k] + nz1;
                        bool skip_x = !pbc[0] && (nx2 < 0 || nx2 >= N_cell[0]);
                        bool skip_y = !pbc[1] && (ny2 < 0 || ny2 >= N_cell[1]);
                        bool skip_z = !pbc[2] && (nz2 < 0 || nz2 >= N_cell[2]);
                        if (skip_x || skip_y || skip_z) continue;

                        neighbor_list[neighbor_index] = find_index
                        (
                            nx2, ny2, nz2, N_cell[0], N_cell[1], N_cell[2], 
                            hopping_data[3][k], N_orbital
                        );

                        real x12 = lattice_constant[transport_direction]
                                   * hopping_data[transport_direction][k];
                        x12 += x_cell[hopping_data[3][k]] - x_cell[m];
                        xx[neighbor_index] = x12;

                        hopping_real[neighbor_index] = hopping_data[4][k]; 
                        hopping_imag[neighbor_index] = hopping_data[5][k]; 

                        ++count;
                    } 
                    neighbor_number[n1] = count;  
                }
            }
        }
    }
    print_finished_reading(filename);
}




