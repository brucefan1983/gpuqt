#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/

 
#include "vector.h"
#include "model.h"


/*
	Gets number of elements from model and sets array and grid sizes accordingly.
	Allocates memory on the device
*/
void Vector::initialize_parameters()
{
    n = model.number_of_atoms;
    array_size = n * sizeof(real);
    grid_size = (n-1) / BLOCK_SIZE + 1;
    hipMalloc((void**)&real_part, array_size);
    hipMalloc((void**)&imag_part, array_size);	
}


/*
	Kernel for setting all elements of a state to zero (both real and imaginary parts)
*/
__global__ void gpu_set_zero(int number_of_elements, real *g_state_real, real *g_state_imag)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < number_of_elements)
    {
        g_state_real[n] = 0; 
        g_state_imag[n] = 0;  
    }
}


/*
	Constructor for an empty vector. 
	Takes length from model and sets all elements to zero
*/
Vector::Vector(Model& model) : model(model)
{
    initialize_parameters();	
    gpu_set_zero<<<grid_size, BLOCK_SIZE>>>(n, real_part, imag_part);
}



/*
	Constructor for a vector of arbitrary length. 
	Does not initialize data. 
*/
Vector::Vector(int n, Model& model) : model(model)
{
    this->n = n;
    array_size = n * sizeof(real);
    grid_size = (model.number_of_atoms-1) / BLOCK_SIZE + 1;
    hipMalloc((void**)&real_part, array_size);
    hipMalloc((void**)&imag_part, array_size);
}



/*
	Kernel for copying states on the gpu
*/
__global__ void gpu_copy_state
(int number_of_atoms, real *g_state_in_real, real *g_state_in_imag, real *g_state_out_real, real *g_state_out_imag)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < number_of_atoms)
    {
        g_state_out_real[n] = g_state_in_real[n]; 
        g_state_out_imag[n] = g_state_in_imag[n];  
    }
}


/*
	Constructor which creates a copy of *original*
*/
Vector::Vector(Vector& original) : model(original.model)
{
    initialize_parameters();
    gpu_copy_state<<<grid_size, BLOCK_SIZE>>>(n, original.real_part, original.imag_part, real_part, imag_part);
}




// Destructor
Vector::~Vector()
{
    hipFree(real_part);
    hipFree(imag_part);
}


// Add the "other" vector to the current vector. This is the kernel
__global__ void gpu_add_state
(int n, real *g_state_in_real, real *g_state_in_imag, real *g_state_out_real, real *g_state_out_imag)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        g_state_out_real[i] += g_state_in_real[i]; 
        g_state_out_imag[i] += g_state_in_imag[i];  
    }
}


// Add the "other" vector to the current vector. This is a wrapper function
void Vector::add(Vector& other, real coeff)
{
    gpu_add_state<<<grid_size, BLOCK_SIZE>>>(n, other.real_part, other.imag_part, real_part, imag_part);
}


// Sets this vector to the same state as "other" vector
void Vector::copy(Vector& other)
{
    if (other.n == n)
    {
        gpu_copy_state<<<grid_size, BLOCK_SIZE>>>(n, other.real_part, other.imag_part, real_part, imag_part);	
    }
    else
    {
        std::cout << "Array sizes do not match in copy." << std::endl;	
    }
}


// Copy from a host vector to the current vector
void Vector::copy_from_host(real* other_real, real* other_imag)
{
    hipMemcpy(real_part, other_real, array_size, hipMemcpyHostToDevice);
    hipMemcpy(imag_part, other_imag, array_size, hipMemcpyHostToDevice);		
}


// Copy the current vector to a host vector
void Vector::copy_to_host(real* target_real, real* target_imag)
{
    hipMemcpy(target_real, real_part, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(target_imag, imag_part, array_size, hipMemcpyDeviceToHost);
}


// Exchange the pointers of the current vector and the "other" vector
void Vector::swap(Vector& other)
{
    real* tmp_real = real_part;
    real* tmp_imag = imag_part;
    real_part = other.real_part,
    imag_part = other.imag_part;
    other.real_part = tmp_real;
    other.imag_part = tmp_imag;
}


/*
	Device function which performs sum reduction over warp
*/
__device__ void warp_reduce(volatile real *s, int t)
{
    s[t] += s[t + 32]; s[t] += s[t + 16]; s[t] += s[t + 8];
    s[t] += s[t + 4];  s[t] += s[t + 2];  s[t] += s[t + 1];
}


// The first step of calculating the inner products. This is the kernel
__global__ void gpu_find_inner_product_1
(
    int number_of_atoms,
    real *g_final_state_real, 
    real *g_final_state_imag, 
    real *g_random_state_real,
    real *g_random_state_imag,
    real *g_inner_product_real, 
    real *g_inner_product_imag,
    int   g_offset
)
{
    int tid = threadIdx.x;
    int n = blockIdx.x * blockDim.x + tid;
    int m;
    real a, b, c, d;
    __shared__ real s_data_real[BLOCK_SIZE];
    __shared__ real s_data_imag[BLOCK_SIZE];
    s_data_real[tid] = 0.0;
    s_data_imag[tid] = 0.0;
    
    if (n < number_of_atoms)
    {
        a = g_final_state_real[n];
        b = g_final_state_imag[n];
        c = g_random_state_real[n];
        d = g_random_state_imag[n];
        s_data_real[tid] = (a * c + b * d); 
        s_data_imag[tid] = (b * c - a * d);
    }
    __syncthreads();

    if (tid < 256) {m = tid + 256; s_data_real[tid] += s_data_real[m]; s_data_imag[tid] += s_data_imag[m];}
    __syncthreads();
    if (tid < 128) {m = tid + 128; s_data_real[tid] += s_data_real[m]; s_data_imag[tid] += s_data_imag[m];}
    __syncthreads();
    if (tid < 64)  {m = tid + 64;  s_data_real[tid] += s_data_real[m]; s_data_imag[tid] += s_data_imag[m];}
    __syncthreads();
    if (tid < 32)  {warp_reduce(s_data_real, tid); warp_reduce(s_data_imag, tid);}
    if (tid == 0) 
    {        
        g_inner_product_real[blockIdx.x + g_offset] = s_data_real[0];
        g_inner_product_imag[blockIdx.x + g_offset] = s_data_imag[0];
    }
}


// The first step of calculating the inner products. This is a wrapper function
void Vector::inner_product_1(Vector& other, Vector& target, int offset)
{
    gpu_find_inner_product_1<<<grid_size, 512>>>
    (
        model.number_of_atoms, real_part, imag_part, 
        other.real_part, other.imag_part, target.real_part, target.imag_part, 
        offset
    );
}


// The second step of calculating the inner products. This is the kernel
__global__ void gpu_find_inner_product_2
(
    int number_of_atoms,	
    real *g_inner_product_1_real, 
    real *g_inner_product_1_imag,
    real *g_inner_product_2_real, 
    real *g_inner_product_2_imag
)
{
    //<<<para.number_of_energy_points, BLOCK_SIZE)>>>
    int tid = threadIdx.x;
    int patch, n, m;

    __shared__ real s_data_real[BLOCK_SIZE];
    __shared__ real s_data_imag[BLOCK_SIZE];
    s_data_real[tid] = 0.0;
    s_data_imag[tid] = 0.0;
    int number_of_blocks  = (number_of_atoms - 1) / BLOCK_SIZE + 1;
    int number_of_patches = (number_of_blocks - 1) / BLOCK_SIZE + 1;

    for (patch = 0; patch < number_of_patches; ++patch)
    {
        n = tid + patch * BLOCK_SIZE;
        if (n < number_of_blocks)
        {
            m = blockIdx.x * number_of_blocks + n;
            s_data_real[tid] += g_inner_product_1_real[m]; 
            s_data_imag[tid] += g_inner_product_1_imag[m];
        }
    }
    __syncthreads();
  
    if (tid < 256) {m = tid + 256; s_data_real[tid] += s_data_real[m]; s_data_imag[tid] += s_data_imag[m];}
    __syncthreads();
    if (tid < 128) {m = tid + 128; s_data_real[tid] += s_data_real[m]; s_data_imag[tid] += s_data_imag[m];}
    __syncthreads();
    if (tid < 64) {m = tid + 64; s_data_real[tid] += s_data_real[m]; s_data_imag[tid] += s_data_imag[m];}
    __syncthreads();
    if (tid < 32) {warp_reduce(s_data_real, tid); warp_reduce(s_data_imag, tid);}
    if (tid == 0) 
    {        
        g_inner_product_2_real[blockIdx.x] = s_data_real[0];
        g_inner_product_2_imag[blockIdx.x] = s_data_imag[0];
    }    
}


// The second step of calculating the inner products. This is a wrapper function
void Vector::inner_product_2(Vector& target)
{
    gpu_find_inner_product_2<<<model.number_of_moments, 512>>>
    (
        model.number_of_atoms, real_part, imag_part, 
        target.real_part, target.imag_part
    );	
}



