#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "lsqt.h"
#include <iostream>
#include <fstream>
using namespace std;


//----------------------------------------------------------------------------80
int main(int argc, char *argv[])
{
    cout << endl;
    cout << "***************************************************************\n";
    cout << "*                  Welcome to use LSQT                        *\n";
    cout << "*          (Linear Scaling Quantum Transport)                 *\n";
    cout << "*        (Author:  Zheyong Fan <brucenju@gmail.com>)          *\n";
    cout << "***************************************************************\n";
    cout << endl;
	
    if (argc != 2)
    {
        cout << "Usage: src/gpuqt input.txt" << std::endl;
        exit(1);
    }
	
    ifstream input(argv[1]); // input = the driver input file
    if (!input.is_open())
    {
        cout << "Failed to open " << argv[1] << endl;
        exit(1);
    }		

    string directory;
    while (getline(input, directory))
    {
        if (directory == "")
             continue;
        cout << endl;
        cout << "===========================================================\n";
        cout << "Run LSQT simulation for " << directory << std::endl; 
        cout << "===========================================================\n";

        clock_t time_begin = clock();
         
        // call the driver function
        lsqt(directory);

        clock_t time_finish = clock();
        double time_used = double(time_finish - time_begin) / CLOCKS_PER_SEC;

        cout << endl;
        cout << "===========================================================\n";
        cout << "Total time used for " << directory << " = " 
             << time_used <<" s" << endl; 
        cout << "===========================================================\n";
    }

    return 0;
}


