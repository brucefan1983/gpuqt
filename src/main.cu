#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
    The main function of the LSQT code
------------------------------------------------------------------------------*/

#include "lsqt.h"
#include <fstream>
#include <iostream>
using namespace std;

static void print_welcome();
static void check_argc(int);
static void print_start(std::string);
static void print_finish(std::string, real);

int main(int argc, char* argv[])
{
  print_welcome();
  check_argc(argc);
  ifstream input(argv[1]); // input = the driver input file
  if (!input.is_open()) {
    cout << "Failed to open " << argv[1] << endl;
    exit(1);
  }
  string directory;
  while (getline(input, directory)) {
    if (directory == "") {
      continue;
    }
    print_start(directory);
    clock_t time_begin = clock();
    lsqt(directory);
    clock_t time_finish = clock();
    real time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
    print_finish(directory, time_used);
  }
  return 0;
}

static void print_welcome()
{
  cout << endl;
  cout << "***************************************************************\n";
  cout << "*                  Welcome to use LSQT                        *\n";
  cout << "*          (Linear Scaling Quantum Transport)                 *\n";
  cout << "*        (Author:  Zheyong Fan <brucenju@gmail.com>)          *\n";
  cout << "***************************************************************\n";
  cout << endl;
}

static void check_argc(int argc)
{
  if (argc != 2) {
    cout << "Usage: src/gpuqt input.txt" << std::endl;
    exit(1);
  }
}

static void print_start(std::string directory)
{
  cout << endl;
  cout << "===============================================================\n";
  cout << "Run LSQT simulation for " << directory << std::endl;
  cout << "===============================================================\n";
}

static void print_finish(std::string directory, real time)
{
  cout << endl;
  cout << "===============================================================\n";
  cout << "Total time used for " << directory << " = " << time << " s" << endl;
  cout << "===============================================================\n";
}
