#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "common.h"
#include "gpuqt.h"




//----------------------------------------------------------------------------80
int main(int argc, char *argv[])
{	
    if (argc != 2)
    {
        std::cout << "Usage: src/gpuqt input.txt" << std::endl;
        exit(1);
    }
	
    std::ifstream input(argv[1]); // input = the driver input file
    if (!input.is_open())
    {
        std::cout << "Failed to open " << argv[1] << std::endl;
        exit(1);
    }		

    std::string directory;
    while (std::getline(input, directory))
    {
        if (directory == "")
             continue;
        std::cout << std::endl;
        std::cout << "===========================================" << std::endl;
        std::cout << "Run KGQT simulation for " << directory << std::endl; 
        std::cout << "===========================================" << std::endl;

        clock_t time_begin = clock();
         
        // call the driver function
        gpuqt(directory);

        clock_t time_finish = clock();
        double time_used = double(time_finish - time_begin) / CLOCKS_PER_SEC;

        std::cout << std::endl;
        std::cout << "===========================================" << std::endl;
        std::cout << "Total time used for " << directory << " = " 
                  << time_used <<" s" << std::endl; 
        std::cout << "===========================================" << std::endl;
    }

    return 0;
}




