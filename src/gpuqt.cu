#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "gpuqt.h"
#include "vector.h"
#include "hamiltonian.h"
#include "sigma.h"
#include "model.h"




void gpuqt(std::string input_directory)
{
    // Initialize model on the CPU
    Model model(input_directory);
    Hamiltonian H(model);
    Vector random_state(model);

    clock_t time_begin, time_finish;
    real time_used;
    
    // Loop over different random vectors
    for (int i = 0; i < model.number_of_random_vectors; ++i)
    {
        std::cout << "Starting vector number " << i+1 << std::endl;
        model.initialize_state(random_state);

        // Always calculate the DOS, since it is very cheap
        time_begin = clock(); 
        find_dos(model, H, random_state);
        time_finish = clock();
        time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
        std::cout << "Time used for finding DOS = " 
                  << time_used << " s" << std::endl; 

        // Calculate the MSD only if you want to
        if (model.calculate_msd == 1)  
        {    
            time_begin = clock();
            find_msd(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
            std::cout << "Time used for finding MSD = " 
                      << time_used << " s" << std::endl;
        }

        // Calculate the VAC only if you want to
        if (model.calculate_vac == 1)  
        {
            time_begin = clock();
            find_vac(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
            std::cout << "Time used for finding VAC = " 
                      << time_used << " s" << std::endl;
        }
    }  
      	
}



