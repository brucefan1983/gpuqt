#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan, Ville Vierimaa, and Ari Harju

    This file is part of GPUQT.

    GPUQT is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUQT is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUQT.  If not, see <http://www.gnu.org/licenses/>.
*/




#include "lsqt.h"
#include "vector.h"
#include "hamiltonian.h"
#include "sigma.h"
#include "model.h"
#include <iostream>
typedef double real;




static void print_started_random_vector(int i)
{
    std::cout << std::endl;
    std::cout << "===========================================================";
    std::cout << std::endl;
    std::cout << "Started  simulation with random vector number " 
              << i << std::endl;
    std::cout << std::endl;
}



static void print_finished_random_vector(int i)
{
    std::cout << std::endl;
    std::cout << "Finished simulation with random vector number " 
              << i << std::endl; 
    std::cout << "===========================================================";
    std::cout << std::endl << std::endl;
}




void lsqt(std::string input_directory)
{
    // Initialize model on the CPU
    Model model(input_directory);
    Hamiltonian H(model);
    Vector random_state(model.number_of_atoms);

    clock_t time_begin, time_finish;
    real time_used;

    // Loop over different random vectors
    for (int i = 0; i < model.number_of_random_vectors; ++i)
    {
        print_started_random_vector(i);

        int orbital = -1; // using random vectors rather than a local orbital
        model.initialize_state(random_state, orbital);

        // Always calculate the DOS, since it is very cheap
        time_begin = clock(); 
        find_dos(model, H, random_state, 0);
        time_finish = clock();
        time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
        std::cout << "- Time used for finding DOS = " 
                  << time_used << " s" << std::endl; 

        // Calculate the MSD only if you want to
        if (model.calculate_msd == 1)  
        {    
            time_begin = clock();
            find_msd(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
            std::cout << "- Time used for finding MSD = " 
                      << time_used << " s" << std::endl;
        }

        // Calculate the VAC only if you want to
        if (model.calculate_vac == 1)  
        {
            time_begin = clock();
            find_vac(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
            std::cout << "- Time used for finding VAC = " 
                      << time_used << " s" << std::endl;
        }

        // Calculate the spin polarization only if you want to
        if (model.calculate_spin == 1)  
        {
            time_begin = clock();
            find_spin_polarization(model, H, random_state);
            time_finish = clock();
            time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
            std::cout << "- Time used for finding spin polarization = " 
                      << time_used << " s" << std::endl;
        }

        print_finished_random_vector(i);
    }

    // Calculate the LDOS only if you want to
    if (model.calculate_ldos)
    {
        time_begin = clock();
        // loop over the local orbitals
        for (int i = 0; i < model.number_of_local_orbitals; ++i)
        {
            int orbital = model.local_orbitals[i];
            model.initialize_state(random_state, orbital);
            find_dos(model, H, random_state, orbital);
            std::cout << "- Finished orbital " << orbital << std::endl;
        }
        time_finish = clock();
        time_used = real(time_finish - time_begin) / CLOCKS_PER_SEC;
        std::cout << "- Time used for finding LDOS = "
                  << time_used << " s" << std::endl;
    }
}



